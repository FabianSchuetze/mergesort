#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <sys/time.h>
#include "../include/merge.h"
#define CHECK(call)                                                \
    {                                                              \
        const hipError_t error = call;                            \
        if (error != hipSuccess) {                                \
            fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
            fprintf(stderr, "code: %d, reason: %s\n", error,       \
                    hipGetErrorString(error));                    \
        }                                                          \
    }

__device__ void merge_sequential(int* A, int m, int* B, int n, int* C) {
    int i = 0;  // index A
    int j = 0;  // index B
    int k = 0;  // index C

    while ((i < m) && (j < n)) {
        if (A[i] <= B[j]) {
            C[k++] = A[i++];
        } else {
            C[k++] = B[j++];
        }
    }
    if (i == m) {
        // done with A[] handle remaining B
        for (; j < n; j++) {
            C[k++] = B[j];
        }
    } else {
        for (; i < m; i++) {
            // done with B[] handle remaining A
            C[k++] = A[i];
        }
    }
}

//Identifies location in A for range of merging
__device__ void co_rank(int k, const int* A, int m, const int* B, int n, 
                        int* out) {
    int i = k < m ? k : m;
    int j = k - i;
    int i_low = 0 > (k - n) ? 0 : k - n;
    int j_low = 0 > (k - m) ? 0 : k - m;
    int delta;
    bool active = true;
    while (active) {
        if (i > 0 && j < n && A[i - 1] > B[j]) {
            delta = ((i - i_low + 1) >> 1);
            j_low = j;
            j = j + delta;
            i = i - delta;
        } else if (j > 0 && i < m && B[j - 1] >= A[i]) {
            delta = ((j - j_low + 1) >> 1);
            i_low = i;
            i = i + delta;
            j = j - delta;
        } else {
            active = false;
        }
    }
    out[0] = i;
}

__global__ void merge_basic_kernel(int* A, int m, int* B, int n, int* C) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = m + n;
    int k_curr = tid * ceilf((sum) / (blockDim.x * gridDim.x));
    int k_next = min((tid + 1) * ceilf(sum / (blockDim.x * gridDim.x)), sum);
    int i_curr;
    int i_next;
    co_rank(k_curr, A, m, B, n, &i_curr);
    co_rank(k_next, A, m, B, n, &i_next);
    int j_curr = k_curr - i_curr;
    int j_next = k_next - i_next;
    //printf(
        //"k_curr %d, k_next %i "
        //"i_curr %i, i_next %i, j_curr %i, j_next %i, tid %d\n",
        //k_curr, k_next, i_curr, i_next, j_curr, j_next, tid);
    merge_sequential(&A[i_curr], i_next - i_curr, &B[j_curr], j_next - j_curr,
                     &C[k_curr]);
}

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1e-6);
}

void cuda_merge(const int* A, int m, const int* B, int n, int* C) {
    int* d_A;
    int* d_B;
    int* d_C;
    CHECK(hipMalloc((void**)&d_A, m * sizeof(int)));
    CHECK(hipMalloc((void**)&d_B, n * sizeof(int)));
    CHECK(hipMalloc((void**)&d_C, (m + n) * sizeof(int)));
    CHECK(hipMemcpy(d_A, A, m * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, B, n * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_C, C, (m + n) * sizeof(int), hipMemcpyHostToDevice));
    dim3 blockDim(4);
    dim3 gridDim(128); //ten threads, likely bug is too many selected
    double cpuStart = cpuSecond();
    merge_basic_kernel<<<blockDim, gridDim>>>(d_A, m, d_B, n, d_C);
    CHECK(hipDeviceSynchronize());
    double cpuEnd = cpuSecond() - cpuStart;
    printf("The GPU took %.7f\n", cpuEnd);
    CHECK(hipMemcpy(C, d_C, (m + n) * sizeof(int), hipMemcpyDeviceToHost));
}
