#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include "../include/common.h"
#include "../include/merge.h"

#include "../include/merge.h"

__device__ int mergepath(const int* a, int size_a, const int* b, int size_b,
                         int diag) {
    if (diag == 0) return 0;
    int begin = max(0, diag - size_b);
    int end = min(diag, size_a);

    while (begin < end) {
        int mid = (begin + end) >> 1;
        int a_val = a[mid];
        int b_val = b[diag - 1 - mid];
        bool pred = a_val < b_val;
        if (pred)
            begin = mid + 1;
        else
            end = mid;
    }
    return begin;
}

__device__ void merge(const int* a, int start_a, int sz_a, const int* b,
                      int start_b, int sz_b, int* c, int start_c, int length) {
    int i = 0;
    int j = 0;
    int k = 0;
    while (k < length) {
        if (start_a + i == sz_a)
            c[start_c + k++] = b[start_b + j++];
        else if (start_b + j == sz_b)
            c[start_c + k++] = a[start_a + i++];
        else if (a[start_a + i] <= b[start_b + j])
            c[start_c + k++] = a[start_a + i++];
        else
            c[start_c + k++] = b[start_b + j++];
    }
}

__global__ void paralleMerge(const int* a, int sz_a, const int* b, int sz_b,
                             int* c, int length) {
    /*int process = */
    int process = blockIdx.x * blockDim.x + threadIdx.x;
    int diag = process * length;
    int a_start = mergepath(a, sz_a, b, sz_b, diag);
    int b_start = diag - a_start;
    merge(a, a_start, sz_a, b, b_start, sz_b, c, diag, length);
}

__global__ void cuda_determine_range(const int* a, int sz_a, const int* b,
                                     int sz_b, int shared_size,
                                     int* block_boundaries) {
    int diag = shared_size * threadIdx.x;
    printf("the diag is %i, at thread %i with blockDim %i\n", diag, threadIdx.x,
           blockDim.x);
    if (diag < sz_a + sz_b) {
        int a_start = mergepath(a, sz_a, b, sz_b, diag);
        block_boundaries[threadIdx.x] = a_start;
        block_boundaries[threadIdx.x + blockDim.x + 1] = diag - a_start;
        printf("The range for thread %i at blockDim %i is %i\n", threadIdx.x,
               blockDim.x, a_start);
    }
}
void determine_range(const int* d_A, int sz_a, const int* d_B, int sz_b,
                     int size_shared, const int* boundaries,
                     int* d_boundaries) {
    int n_blocks = ceilf((sz_a + sz_b) / size_shared);
    hipMalloc((void**)&d_boundaries, 2 * n_blocks * sizeof(int));
    hipMemcpy(d_boundaries, boundaries, 2 * n_blocks * sizeof(int),
               hipMemcpyHostToDevice);
    cuda_determine_range<<<1, gridDim>>>(d_A, sz_a, d_B, sz_b, size_shared,
                                         d_boundaries);
}
void cuda_merge(const int* d_A, int sz_a, const int* d_B, int sz_b, int* d_C,
                int length) {
    dim3 blockDim(10);
    int size_shared = 10;
    int n_blocks = ceilf((sz_a + sz_b) / size_shared);
    dim3 gridDim(n_blocks);  // ten threads, likely bug is too many selected
    int boundaries[2 * n_blocks + 2];
    int* d_boundaries;
    determine_range(d_A, sz_a, d_B, sz_b, size_shared, boundaries,
                    d_boundaries);
    // boundaries[n_blocks] = sz_a;
    // boundaries[2 * n_blocks] = sz_b;
    // hipMemcpy(boundaries, d_boundaries, 2 * n_blocks * sizeof(int),
    // hipMemcpyDeviceToHost);
    // MY_CHECK(hipDeviceSynchronize());
    for (int i = 0; i < n_blocks; ++i) {
        printf("At block %i the range of a is (%i, %i) and of b is (%i, %i)\n",
               i, boundaries[i], boundaries[i + 1], boundaries[i + n_blocks],
               boundaries[i + 1 + n_blocks]);
    }
    paralleMerge<<<gridDim, blockDim>>>(d_A, sz_a, d_B, sz_b, d_C, length,
                                        d_boundaries);
}
