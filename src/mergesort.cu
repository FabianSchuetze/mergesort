#include "hip/hip_runtime.h"


__device__ int mergepath(int* a, int size_a, int* b, int size_b, int diag) {
    if (diag == 0)
        return 0;
    int begin = max(0, diag - size_b);
    int end =min(diag, size_a);

    while (begin < end) {
        int mid = (begin + end) >> 1;
        int a_val = a[mid];
        int b_val = b[diag - 1 - mid];
        bool pred = a_val < b_val;
        if (pred)
            begin = mid + 1;
        else
            end = mid;
    }
    return begin;
}

__device__ void merge(int* a, int start_a, int sz_a, int* b, int start_b, 
                      int sz_b, int* c, int start_c, int length) {
    int i = 0;
    int j = 0;
    int k = 0;
    while (k < length) {
        if (start_a + i == sz_a)
            c[start_c + k++] = b[start_b + j++];
        else if (start_b + j == sz_b)
            c[start_c + k++] = a[start_a + i++];
        else if (a[start_a + i] <= b[start_b + j])
            c[start_c + k++] = a[start_a + i++];
        else
            c[start_c + k++] = b[start_b + j++];
    }
}

__global__ void paralleMerge(int*a, int sz_a, int* b, int sz_b, int* c, 
                             int length) {
    /*int process = */
    int process = blockIdx.x * blockDim.x + threadIdx.x;
    int diag = process * length;
    int a_start = mergepath(a, sz_a, b, sz_b, diag);
    int b_start = diag - a_start;
    merge(a, a_start, sz_a, b, b_start, sz_b, c, diag, length);
}


void merge(int* d_A, int sz_a, int* d_B, int sz_b, int* d_C, int length) {
    dim3 blockDim(1);
    dim3 gridDim(32);  // ten threads, likely bug is too many selected
    paralleMerge<<<blockDim, gridDim>>>(d_A, sz_a, d_B, sz_b,  length);
}
